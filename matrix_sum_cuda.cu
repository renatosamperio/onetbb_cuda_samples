#include "hip/hip_runtime.h"
#include "matrix_utils_cuda.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matrix_sum_kernel(const float* A, const float* B, const float* C, float* result, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // printf("  Executing kernel on (%d, %d)", idx, idy);
    if (idx < N && idy < N) {
        int index = idy * N + idx;
        result[index] = A[index] + B[index] + C[index];
    }
}

__global__ void matrix_addition(const float* A, const float* B, float* C, float* result, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        result[index] = A[index] + B[index] + C[index];
    }
}


extern "C"
void matrix_sum_cuda(const float* A, const float* B, const float* C, float* result, int N, int idx) {
    float *d_A, *d_B, *d_C, *d_result;
    size_t size = N * sizeof(float);

    CUDA_CHECK(hipMalloc(&d_A, size));
    CUDA_CHECK(hipMalloc(&d_B, size));
    CUDA_CHECK(hipMalloc(&d_C, size));
    CUDA_CHECK(hipMalloc(&d_result, size));

    CUDA_CHECK(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C, C, size, hipMemcpyHostToDevice));

    // dim3 threadsPerBlock(65536, 65536);
    // dim3 threadsPerBlock(1024, 1024);
    // dim3 threadsPerBlock(16, 16);
    dim3 threadsPerBlock(16, 4);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // std::cout << "    CUDA summing in: " << idx << std::endl;
    matrix_sum_kernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, d_result, N);
    CUDA_CHECK(hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipFree(d_result));
}

extern "C"
void gpu_matrix_addition(const float* A, const float* B, const float* C, float* result, int N) {
    float *d_A, *d_B, *d_C, *d_result;
    size_t size = N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    hipMalloc(&d_result, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    matrix_addition<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, d_result, N);

    hipMemcpy(result, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}